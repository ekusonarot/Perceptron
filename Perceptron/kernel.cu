
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <ctime>

#include "weight.h"
#include "linear.cuh"

int main() {
	Linear linear("input", 2, 2);
	Linear linear1("output", 2, 1, &linear);
	float input0[2] = { 0,0 }, input1[2] = { 0,1 }, input2[2] = { 1,0 }, input3[2] = { 1,1 };
	float target0[1] = { 1 }, target1[1] = { 0 }, target2[1] = { 0 }, target3[1] = { 1 };
	for (int i = 0; i < 100000; i++) {
		linear.setInput(input0);
		linear.forward();
		linear1.forward();
		linear1.backward(target0);
		linear.backward();
		if (i % 1001 == 0)
			std::cout << input0[0] << " " << input0[1] << " " << "=" << " " << linear1.getOutput()[0] << std::endl;

		linear.setInput(input1);
		linear.forward();
		linear1.forward();
		linear1.backward(target1);
		linear.backward();
		if (i % 1001 == 0)
			std::cout << input1[0] << " " << input1[1] << " " << "=" << " " << linear1.getOutput()[0] << std::endl;

		linear.setInput(input2);
		linear.forward();
		linear1.forward();
		linear1.backward(target2);
		linear.backward();
		if (i % 1001 == 0)
			std::cout << input2[0] << " " << input2[1] << " " << "=" << " " << linear1.getOutput()[0] << std::endl;

		linear.setInput(input3);
		linear.forward();
		linear1.forward();
		linear1.backward(target3);
		linear.backward();
		if (i % 1001 == 0)
			std::cout << input3[0] << " " << input3[1] << " " << "=" << " " << linear1.getOutput()[0] << std::endl;
	}


	linear.setInput(input0);
	linear.forward();
	linear1.forward();
	std::cout << input0[0] << " " << input0[1] << " " << "=" << " " << linear1.getOutput()[0] << std::endl;

	linear.setInput(input1);
	linear.forward();
	linear1.forward();
	std::cout << input1[0] << " " << input1[1] << " " << "=" << " " << linear1.getOutput()[0] << std::endl;

	linear.setInput(input2);
	linear.forward();
	linear1.forward();
	std::cout << input2[0] << " " << input2[1] << " " << "=" << " " << linear1.getOutput()[0] << std::endl;

	linear.setInput(input3);
	linear.forward();
	linear1.forward();
	std::cout << input3[0] << " " << input3[1] << " " << "=" << " " << linear1.getOutput()[0] << std::endl;
	
	linear.saveWeight();
	linear1.saveWeight();
}